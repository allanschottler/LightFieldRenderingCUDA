#include "hip/hip_runtime.h"
/* 
 * File:   LightFieldRender.cu.cpp
 * Author: allan
 */

#include <cmath>
#include <iostream>
#include <stdio.h>

#include "LightFieldRender.h"
#include "hip/hip_vector_types.h"
#include "CUDAManager.h"

__constant__ LightFieldRender::KernelParameters _cudaKernelParameters;

texture< uchar4, hipTextureType2D, hipReadModeNormalizedFloat > _lightfieldTexture;

struct Ray
{
    float3 origin;
    float3 direction;
};

struct Quad
{
    float3 tlPoint; // top left s1
    float3 trPoint; // top right s2
    float3 blPoint; // bottom left s3
};

struct Plane
{
    float3 normal;
    float3 point;
};

// Auxiliares
__device__
bool intersectPlane( Ray ray, Plane plane, float3* intersectedPoint )
{    
    // Testa se acerta plano
    float ndotdR = dot( plane.normal, ray.direction );

    if( fabs( ndotdR ) < 1e-6f ) // tolerance
        return false;
    
    // Calcula ponto de interseção
    float t = dot( -plane.normal, ( ray.origin - plane.point ) ) / ndotdR;
    *intersectedPoint = ray.origin + ray.direction * t;
    
    return t >= 0;
}

__device__
bool intersectQuad( Ray ray, Quad quad, float3* intersectedPoint )
{
    // Define normal
    float3 dS21 = quad.trPoint - quad.tlPoint;
    float3 dS31 = quad.blPoint - quad.tlPoint;
    float3 n = cross( dS21, dS31 );
    
    Plane plane;
    plane.normal = n;
    plane.point = quad.tlPoint;

    if( intersectPlane( ray, plane, intersectedPoint ) )
    {
        // Verifica se interseção está contido no quad
        float3 dMS1 = *intersectedPoint - quad.tlPoint; 
        float u = dot( dMS1, dS21 );
        float v = dot( dMS1, dS31 );

        return ( u >= 0.0f && u <= dot( dS21, dS21 ) && v >= 0.0f && v <= dot( dS31, dS31 ) );
    }
    
    return false;
}

__device__
unsigned int rgbaFloatToInt( float4 rgba )
{
    rgba.x = __saturatef( rgba.x ); 
    rgba.y = __saturatef( rgba.y );
    rgba.z = __saturatef( rgba.z );
    rgba.w = __saturatef( rgba.w );

    typedef unsigned int uint;

    return ( uint( rgba.w * 255 ) << 24 ) | ( uint( rgba.z * 255 ) << 16 ) |
           ( uint( rgba.y * 255 ) << 8 ) | uint( rgba.x * 255 );  
}

__device__
inline float map( float x, float in_min, float in_max, float out_min, float out_max )
{
    return ( x - in_min ) * ( out_max - out_min ) / ( in_max - in_min ) + out_min;
}

__device__
void setNearAndFar( float u, float v, float& nearX, float& farX, float& nearY, float& farY, float& nearZ, float& farZ )
{
    // Calcula os pontos extremos do raio em coordenadas de mundo.
    nearX = _cudaKernelParameters.nearOrigin.x + u * _cudaKernelParameters.uNear.x + v * _cudaKernelParameters.vNear.x;
    nearY = _cudaKernelParameters.nearOrigin.y + u * _cudaKernelParameters.uNear.y + v * _cudaKernelParameters.vNear.y;
    nearZ = _cudaKernelParameters.nearOrigin.z + u * _cudaKernelParameters.uNear.z + v * _cudaKernelParameters.vNear.z;

    farX = _cudaKernelParameters.farOrigin.x + u * _cudaKernelParameters.uFar.x + v * _cudaKernelParameters.vFar.x;
    farY = _cudaKernelParameters.farOrigin.y + u * _cudaKernelParameters.uFar.y + v * _cudaKernelParameters.vFar.y;
    farZ = _cudaKernelParameters.farOrigin.z + u * _cudaKernelParameters.uFar.z + v * _cudaKernelParameters.vFar.z;
}

__device__
void getNeighbourCameras( float3 hitPoint, float3& c0, float3& c1, float3& c2, float3& c3 )
{    
    c0.x = floor( hitPoint.x );/// sStep );
    c0.y = floor( hitPoint.y );/// tStep );
        
    c1.x = c0.x + 1;
    c1.y = c0.y;
        
    c2.x = c0.x;
    c2.y = c0.y + 1;
        
    c3.x = c0.x + 1;
    c3.y = c0.y + 1;
}

__device__
void mapWorldToTexCoord( Plane& focalPlane, float3 fg, float3 st, float2& uv )
{    
    float sStep = 1. / (_cudaKernelParameters.nCameraCollumns);
    float tStep = 1. / (_cudaKernelParameters.nCameraRows);  
    
    float3 stProj = make_float3( st.x, st.y, focalPlane.point.z );
    float3 stProjToFg = fg - stProj;
    float distanceToFocalPlane = length( stProj - st );
    float3 factor = stProjToFg / distanceToFocalPlane;
    
    uv.x = ( st.x + .5 + factor.x ) * sStep;
    uv.y = ( st.y + .5 + factor.y ) * tStep;    
}

__device__
void trace( Ray& ray, Quad& quad, Plane& focalPlane, float3& hitPoint, float4& collectedColor )
{    
    float3 hitPointPlane;
    bool hit = intersectPlane( ray, focalPlane, &hitPointPlane );
    
    if( !hit ) 
        return; // Nunca deve cair aqui        
    
    float3 c0 = make_float3( 0., 0., 0. );
    float3 c1 = make_float3( 0., 0., 0. );
    float3 c2 = make_float3( 0., 0., 0. );
    float3 c3 = make_float3( 0., 0., 0. );
    
    getNeighbourCameras( hitPoint, c0, c1, c2, c3 );
    
    float2 uv0, uv1, uv2, uv3;
    mapWorldToTexCoord( focalPlane, hitPointPlane, c0, uv0 );
    mapWorldToTexCoord( focalPlane, hitPointPlane, c1, uv1 );
    mapWorldToTexCoord( focalPlane, hitPointPlane, c2, uv2 );
    mapWorldToTexCoord( focalPlane, hitPointPlane, c3, uv3 );
    
    float4 colorA0, colorA1, colorA2, colorA3;
    colorA0 = tex2D( _lightfieldTexture, uv0.x, uv0.y );
    colorA1 = tex2D( _lightfieldTexture, uv1.x, uv1.y );
    colorA2 = tex2D( _lightfieldTexture, uv2.x, uv2.y );
    colorA3 = tex2D( _lightfieldTexture, uv3.x, uv3.y );
    
    collectedColor = lerp( lerp( lerp( colorA0, colorA1, .5 ), colorA2, .5 ), colorA3, .5 );
}

__global__
void d_render( uint* d_output, float* d_depthBuffer, int canvasWidth, int canvasHeight, bool isDebugOn )
{
    //Origem do raio em coordenadas de tela.
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if( x >= canvasWidth || y >= canvasHeight )
        return;

    //Mapeia x e y para o intervalo [-1, 1]
    float u = ( x / ( float ) canvasWidth ) * 2.0f - 1.0f;
    float v = ( y / ( float ) canvasHeight ) * 2.0f - 1.0f;

    float nearX, farX;
    float nearY, farY;
    float nearZ, farZ;
    setNearAndFar( u, v, nearX, farX, nearY, farY, nearZ, farZ );
    
    float3 farPoint = make_float3( farX, farY, farZ );
            
    // Cria o quad
    Quad quad;
    quad.tlPoint = make_float3( 0, 0, 0 );
    quad.blPoint = make_float3( 0, _cudaKernelParameters.nCameraRows - 1, 0 );
    quad.trPoint = make_float3( _cudaKernelParameters.nCameraCollumns - 1, 0, 0 );
    
    Ray eyeRay;
    eyeRay.origin = make_float3( nearX, nearY, nearZ );
    eyeRay.direction = farPoint - eyeRay.origin;
    
    Plane focalPlane;
    focalPlane.normal = normalize( cross( quad.blPoint, quad.trPoint ) );
    focalPlane.point  = make_float3( 0, 0, _cudaKernelParameters.focalPlane );
    
    // Acha a interseção com a bounding box
    float3 hitPoint;
    float4 collectedColor = make_float4( 0, 0, 0, 0 );
        
    bool hit = intersectQuad( eyeRay, quad, &hitPoint );
    
    // Para se o raio não interceptou o quad.
    if( !hit )
        return;
    
    // Traça o raio    
    trace( eyeRay, quad, focalPlane, hitPoint, collectedColor );

    // Grava saidas
    float depth = length( hitPoint - eyeRay.origin ) / length( farPoint - eyeRay.origin );
    
    d_depthBuffer[ y * canvasWidth + x ] = depth;    
    d_output[ y * canvasWidth + x ] = rgbaFloatToInt( collectedColor );
}


// LightFieldRender
float LightFieldRender::renderKernel( dim3 gridSize, dim3 blockSize, uint* d_output, float* d_depthBuffer )
{
    // Repassando parametros para o kernel...
    hipEvent_t clock = CUDAManager::getInstance()->startClock();

    d_render <<< gridSize, blockSize >>> ( d_output, d_depthBuffer, _screenWidth, _screenHeight, true );

    return CUDAManager::getInstance()->stopClock( clock );
}


void LightFieldRender::initLightFieldTexture( unsigned char* texels, int width, int height )
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc< uchar4 >();

    //std::cout << "MALLOC\n";
    CUDAManager::getInstance()->collectError( 
        hipMallocArray( &_lightFieldArray, &channelDesc, width, height ) );    
    
    //std::cout << "MEMCPY\n";
    CUDAManager::getInstance()->collectError( 
        hipMemcpy2DToArray( _lightFieldArray, 0, 0, texels, width * sizeof( uchar4 ), 
        width * sizeof( uchar4 ), height, hipMemcpyHostToDevice ) );
        
    // Inicializa os parametros de textura
    _lightfieldTexture.normalized = true;                       
    _lightfieldTexture.filterMode = hipFilterModeLinear;       
    _lightfieldTexture.addressMode[ 0 ] = hipAddressModeClamp; 
    _lightfieldTexture.addressMode[ 1 ] = hipAddressModeClamp; 

    //std::cout << "BIND\n";
    // Associa o array a textura
    CUDAManager::getInstance()->collectError( 
        hipBindTextureToArray( _lightfieldTexture, _lightFieldArray, channelDesc ) );
}


void LightFieldRender::initKernelParameters()
{    
    //std::cout << "PARAMS\n";
    CUDAManager::getInstance()->collectError(
        hipMemcpyToSymbol(HIP_SYMBOL( _cudaKernelParameters), ( void* ) &_kernelParameters, sizeof( KernelParameters ) ) );
}